#include "hip/hip_runtime.h"
#include <utils.h>

void cudaDeviceInit()
{
    int devCount;
    hipGetDeviceCount(&decCount);
    if(devCount == 0){
        printf("No CUDA capable dices detected.\n");
        exit(EXIT_FAILURE);
    }

    int device = 0;
    bool ok = false;
    while(device < devCount && !ok){
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device);
        if(props.major > 1 || (props.major == 1 && props.minor > 2)){
            ok = true;
        }
        else{
            ++device;
        }
    }
    if(device == devCount){
        printf("No device above 1.2 compute capability detected.\n");
        exit(EXIT_FAILURE);
    }
    else{
        hipSetDevice(device);
    }
}

__global__  void vecAdd(float* A, float* B, float* C)
{
    // threadIdx.x is a built-in variable provided by CUDA at runtime
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

void __cudaSafeCall(hipError_t err, const char *file, int line)
{
    if ((err) != hipSuccess)
    {
        fprintf(stderr, "CUDA error in file %s at line %i: %s.\n", file, line, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
